#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

__global__
void MulMatriz(float* d_min, float* d_mout, int fil, int col)
{
	int i = threadIdx.y + blockDim.y * blockIdx.y;
	int j= threadIdx.x + blockDim.x * blockIdx.x;

	if((i<fil)&&(j<col)) d_mout[i*col+j] = d_min[i*col+j]*5; 
}


int main()
{
	//Inicia reloj ------------------------
	clock_t t_ini, t_fin;
  	double secs;
  	t_ini = clock();
  	//-------------------------------------

	int fil, col;
	float *h_min, *h_mout;
	float *d_min, *d_mout;

	fil = 3;
	col = 4; //con el más grande se hace la referencia para la matriz en 1D

	h_min = (float*)malloc (fil*col*sizeof(float)); //Reserva de memoria en el host
	h_mout = (float*)malloc (fil*col*sizeof(float));

	//Iniciar matriz con valor 13------------------
	for(int i=0; i<fil; i++){
		for(int j=0; j<col; j++){
			h_min[i*col+j] = 13; 
		}
	}

	//Imprimir resultados------------------
	printf("matriz: ----------------------\n"); 

	for(int i=0; i<fil; i++){
		for(int j=0; j<col; j++){
			printf("%f ", h_min[i*col+j]);
		}
		printf("\n"); 
	}	

	printf("\nmatriz x5: ----------------------\n"); 

	//Reserva y copia de datos al dispositivo ---------
	int size = fil*col*sizeof(float);
	int blockSize = 32;
    dim3 dimBlock(blockSize, blockSize, 1);
    dim3 dimGrid(ceil(col/float(blockSize)), ceil(fil/float(blockSize)), 1);
	
	hipMalloc((void **) &d_min, size);
	hipMalloc((void **) &d_mout, size);	
	hipMemcpy(d_min, h_min, size, hipMemcpyHostToDevice);
	//-------------------------------------------------

	MulMatriz<<<dimGrid,dimBlock>>>(d_min, d_mout, fil, col); //Ejecución del kernel

	hipMemcpy(h_mout, d_mout, size, hipMemcpyDeviceToHost); //Copia de datos al host

	//Imprimir resultados------------------
	for(int i=0; i<fil; i++){
		for(int j=0; j<col; j++){
			printf("%f ", h_mout[i*col+j]);
		}
		printf("\n"); 
	}
	//-------------------------------------
	hipFree(d_min); //Liberar memoria del dispositivo
	hipFree(d_mout);
	free(h_min);//Liberar memoria host
	free(h_mout);

	//Fin reloj ------------------------
  	t_fin = clock();
  	secs = (double)(t_fin - t_ini) / CLOCKS_PER_SEC;
  	printf("Tiempo de ejecucion: %.16g milisegundos\n", secs * 1000.0);
  	
  	return 0;
}
