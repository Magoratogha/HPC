#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

__global__ void MulMatriz(float *min, float *mout, int fil, int col)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id < fil*col) 
		mout[id] = min[id]*5; 
}


int main()
{
	//Inicia reloj ------------------------
	clock_t t_ini, t_fin;
  	double secs;
  	t_ini = clock();
  	//-------------------------------------

	int fil, col;
	float *h_min, *h_mout;
	float *d_min, *d_mout;

	fil = 3;
	col = 4; //con el más grande se hace la referencia para la matriz en 1D

	int size = fil*col*sizeof(float); //tamaño en bits de cada matriz

	h_min = (float*)malloc(size);
	h_mout = (float*)malloc(size);
	hipMalloc(&d_min, size);
    hipMalloc(&d_mout, size);

	//Iniciar matriz con valor 13------------------
	for(int i=0; i<fil; i++){
		for(int j=0; j<col; j++){
			h_min[i*col+j] = 13; 
		}
	}

	//Imprimir resultados------------------
	printf("matriz: ----------------------\n"); 
	for(int i=0; i<fil; i++){
		for(int j=0; j<col; j++){
			printf("%f ", h_min[i*col+j]);
		}
		printf("\n"); 
	}	

	printf("\nmatriz x5: ----------------------\n"); 

	hipMemcpy(d_min, h_min, size, hipMemcpyHostToDevice);
	MulMatriz<<<ceil(fil*col/256.0),256>>>(d_min, d_mout, fil, col); //Ejecución del kernel
	hipMemcpy(h_mout, d_mout, size, hipMemcpyDeviceToHost); //Copia de datos al host
	
	//Imprimir resultados------------------
	for(int i=0; i<fil; i++){
		for(int j=0; j<col; j++){
			printf("%f ", h_mout[i*fil+j]);
		}
		printf("\n"); 
	}
	//-------------------------------------

	hipFree(d_min);
    hipFree(d_mout);
	free(h_min);
	free(h_mout);

	//Fin reloj ------------------------
  	t_fin = clock();
  	secs = (double)(t_fin - t_ini) / CLOCKS_PER_SEC;
  	printf("Tiempo de ejecucion: %.16g milisegundos\n", secs * 1000.0);
  	
  	return 0;
}
