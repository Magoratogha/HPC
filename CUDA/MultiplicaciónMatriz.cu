#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__
void MulMatriz(float *d_m, int fil, int col, int n)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if(i<fil*col) d_m[i] = d_m[i]*n; 
}


int main()
{
	//Inicia reloj ------------------------
	clock_t t_ini, t_fin;
  	double secs;
  	t_ini = clock();
  	//-------------------------------------

	int fil, col;
	float* h_in, h_out;
	float* d_m;

	fil = 3;
	col = 4; //con el más grande se hace la referencia para la matriz en 1D

	h_in = (float *)malloc (fil*col*sizeof(float *)); //Reserva de memoria en el host
	h_out = (float *)malloc (fil*col*sizeof(float *));

	//Iniciar matriz con valor 13------------------
	for(int i=0; i<fil; i++){
		for(int j=0; j<col; j++){
			h_in[i*col+j] = 13; 
		}
	}

	//Imprimir resultados------------------
	printf("matriz: ----------------------\n"); 

	for(int i=0; i<fil; i++){
		for(int j=0; j<col; j++){
			printf("%f ", h_in[i*col+j]);
		}
		printf("\n"); 
	}	

	printf("\nmatriz x5: ----------------------\n"); 

	//Reserva y copia de datos al dispositivo ---------
	int size = fil*col*sizeof(float);
	hipMalloc((void **) &d_m, size);
	hipMemcpy(d_m, h_in, size, hipMemcpyHostToDevice);
	//-------------------------------------------------

	MulMatriz<<<ceil(fil*col/256.0),256>>>(d_m, fil, col, 5); //Ejecución del kernel

	hipMemcpy(h_out, d_m, size, hipMemcpyDeviceToHost); //Copia de datos al host
	//Liberar memoria del dispositivo

	//Imprimir resultados------------------
	for(int i=0; i<fil; i++){
		for(int j=0; j<col; j++){
			printf("%f ", h_out[i*fil+j]);
		}
		printf("\n"); 
	}
	//-------------------------------------
	hipFree(d_m);
	free(h_in);
	free(h_out);//Liberar memoria host

	//Fin reloj ------------------------
  	t_fin = clock();
  	secs = (double)(t_fin - t_ini) / CLOCKS_PER_SEC;
  	printf("Tiempo de ejecucion: %.16g milisegundos\n", secs * 1000.0);
  	
  	return 0;
}
