#include<stdio.h>
#include<stdlib.h>
#include<malloc.h>
#include<time.h>
#include<hip/hip_runtime.h>

__global__
void MulMatriz(float* d_Pin, float* d_Pout, int n, int m){
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;

    if ((Row < m) && (Col < n)){
        d_Pout[Row*n+Col] = 2*d_Pin[Row*n+Col]; 
    }
}

__host__
void print(float* M, int rows, int cols){
    printf("-----------MATRIX ------------- \n");
    for(int i=0; i<rows; i++){
        for(int j=0; j<cols; j++){
            printf("%f ", M[i * cols + j]);
        }
        printf("\n");
    }
}

__host__
void receive(float* M, int rows, int cols){
    for(int i=0; i<rows; i++){
        for(int j=0; j<cols; j++){
            M[i*cols+j] = 13;
        }
    }
}

int main(){

    float *A_in, *A_out;
    int rowsA, colsA;    
    
    rowsA = 15;
    colsA = 15;

    //CPU
    A_in = (float*)malloc(rowsA * colsA * sizeof(float));
    A_out = (float*)malloc(rowsA * colsA * sizeof(float));

    receive(A_in, rowsA, colsA);

    //GPU
    hipError_t error = hipSuccess;
    float *d_Ain, *d_Aout;
    int blockSize = 32;
    //int gridSize = ceil((colsA*rowsA) / float(blockSize));
    dim3 dimBlock(blockSize, blockSize, 1);
    dim3 dimGrid(ceil(colsA / float(blockSize)), ceil(rowsA / float(blockSize)), 1);

    error = hipMalloc((void**)&d_Ain, rowsA * colsA * sizeof(float));
    if(error != hipSuccess){
        printf("Error allocating memory d_Ain");
        return 1;
    }

    error = hipMalloc((void**)&d_Aout, rowsA * colsA * sizeof(float));
    if(error != hipSuccess){
        printf("Error allocating memory d_Aout");
        return 1;
    }

    hipMemcpy(d_Ain, A_in, rowsA * colsA * sizeof(float), hipMemcpyHostToDevice);
    MulMatriz<<<dimGrid, dimBlock>>>(d_Ain, d_Aout, rowsA, colsA);
    hipMemcpy(A_out, d_Aout, rowsA * colsA * sizeof(float), hipMemcpyDeviceToHost);

    print(A_out, rowsA, colsA);
    
    free(A_in);
    free(A_out);
    hipFree(d_Ain);
    hipFree(d_Aout);
    return 0;
}