#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

__global__ void MulMatriz(float *m1, float *m2, float *mr, int fil, int col, int w)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int valor;

    if ((i<fil) && (j<col)){
    	valor = 0;
    	for(int k=0; k<w; k++){
    		valor = valor + m1[i*w+k] * m2[k*w+j];
    	}
        mr[i*w+j] = valor; 
    }
}


int main()
{
	//Inicia reloj ------------------------
	clock_t t_ini, t_fin;
  	double secs;
  	t_ini = clock();
  	//-------------------------------------

	int fil1, col1, fil2, col2;
	float *h_m1, *h_m2, *h_mr;
	float *d_m1, *d_m2, *d_mr;

	fil1 = 5;
	col1 = 6; 
	fil2 = 6;
	col2 = 5; 

	int size1 = fil1*col1*sizeof(float); //tamaño en bits de cada matriz
	int size2 = fil2*col2*sizeof(float);
	int sizer = fil1*col2*sizeof(float);

	h_m1 = (float*)malloc(size1);
	h_m2 = (float*)malloc(size2);
	h_mr = (float*)malloc(sizer);
	hipMalloc(&d_m1, size1);
    hipMalloc(&d_m2, size2);
    hipMalloc(&d_mr, sizer);

    int blockSize = 32;
    dim3 dimBlock(blockSize, blockSize, 1);
    dim3 dimGrid(ceil(col2/float(blockSize)), ceil(col2/float(blockSize)), 1);

	//Iniciar matriz 1 con valor 13------------------
	for(int i=0; i<fil1; i++){
		for(int j=0; j<col1; j++){
			h_m1[i*col1+j] = 13; 
		}
	}

	//Iniciar matriz 2 con valor 7------------------
	for(int i=0; i<fil2; i++){
		for(int j=0; j<col2; j++){
			h_m2[i*col2+j] = 7; 
		}
	}

	//Imprimir resultados------------------
	printf("matriz 1: ----------------------\n"); 
	for(int i=0; i<fil1; i++){
		for(int j=0; j<col1; j++){
			printf("%f ", h_m1[i*col1+j]);
		}
		printf("\n"); 
	}

	printf("matriz 2: ----------------------\n"); 
	for(int i=0; i<fil2; i++){
		for(int j=0; j<col2; j++){
			printf("%f ", h_m2[i*col2+j]);
		}
		printf("\n"); 
	}	

	printf("\nmatriz resultado: ----------------------\n"); 

	hipMemcpy(d_m1, h_m1, size1, hipMemcpyHostToDevice);
	hipMemcpy(d_m2, h_m2, size2, hipMemcpyHostToDevice);

	MulMatriz<<<dimGrid, dimBlock>>>(d_m1, d_m2, d_mr, fil1, col2, col1); //Ejecución del kernel
	hipMemcpy(h_mr, d_mr, sizer, hipMemcpyDeviceToHost); //Copia de datos al host
	
	//Imprimir resultados------------------
	for(int i=0; i<fil1; i++){
		for(int j=0; j<col2; j++){
			printf("%f ", h_mr[i*col2+j]);
		}
		printf("\n"); 
	}	
	//-------------------------------------

	hipFree(d_m1);
    hipFree(d_m2);
    hipFree(d_mr);
	free(h_m1);
	free(h_m2);
	free(h_mr);

	//Fin reloj ------------------------
  	t_fin = clock();
  	secs = (double)(t_fin - t_ini) / CLOCKS_PER_SEC;
  	printf("Tiempo de ejecucion: %.16g milisegundos\n", secs * 1000.0);
  	
  	return 0;
}
