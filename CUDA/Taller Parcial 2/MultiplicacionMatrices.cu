#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

__global__ void MulMatriz(float *m1, float *m2, float *mr, int fil1, int col1,int fil2, int col2) {
	int i = blockIdx.y*blockDim.y + threadIdx.y;
	int j = blockIdx.x*blockDim.x + threadIdx.x;
	int valor = 0;

	for (int k=0; k<(blockDim.y+col1-1)/blockDim.y; k++) {
		for (int n=0; n<blockDim.y; ++n)
	    	if ((k*blockDim.y+n<col1 && i<fil1) && (k*blockDim.y+n<fil2 && j<col2))
	      		valor += m1[i*col1+k*blockDim.y+n]*m2[(k*blockDim.y+n)*col2+j];
	}
	if (i<fil1 && j<col2)
		mr[i*col2+j] = valor;
}


__host__
void LeerMatriz(float* m1, float* m2, FILE* file, int fil1, int fil2, int col1, int col2) {
	for(int i=0; i<fil1*col1; i++){
		fscanf(file, "%f", &m1[i]);
    }

	for(int i=0; i<fil2*col2; i++){
		fscanf(file, "%f", &m2[i]);
    }

	fclose(file);
}

__host__
void EscribirMatriz(int fil, int col, float *m) { 
	FILE *f = fopen("output.txt", "a"); 
	for(int i=0; i<fil; i++){
		for(int j=0; j<col-1; j++){
			fprintf(f,"%f,", m[i*col+j]);
		}
		fprintf(f,"%f\n", m[i*col+j-1]); 
	}
	fprintf(f, "\n");
  	fclose(f); 
} 


int main(int argc, char** argv) {
	if (argc != 2) {
        printf("Parametros incorrectos! \n");
        return 1;
    }

	clock_t t_ini, t_fin; //Inicia reloj ------------------------
  	double secs;
  	t_ini = clock();

	int fil1, col1, fil2, col2;
	float *h_m1, *h_m2, *h_mr;
	float *d_m1, *d_m2, *d_mr;

	FILE *archivo;
	archivo = fopen(argv[1], "r");
    fscanf(archivo, "%d %d", &fil1, &col1);
    fscanf(archivo, "%d %d", &fil2, &col2);

	int size1 = fil1*col1*sizeof(float); //tamaño en bits de cada matriz
	int size2 = fil2*col2*sizeof(float);
	int sizer = fil1*col2*sizeof(float);

	h_m1 = (float*)malloc(size1);
	h_m2 = (float*)malloc(size2);
	h_mr = (float*)malloc(sizer);
	hipMalloc(&d_m1, size1);
	hipMalloc(&d_m2, size2);
	hipMalloc(&d_mr, sizer);

	int blockSize = 32;
	dim3 dimBlock(blockSize, blockSize, 1);
	dim3 dimGrid(ceil(col1/float(blockSize)), ceil(col1/float(blockSize)), 1);

	LeerMatriz(h_m1, h_m2, archivo, fil1, fil2, col1, col2); 

	hipMemcpy(d_m1, h_m1, size1, hipMemcpyHostToDevice);
	hipMemcpy(d_m2, h_m2, size2, hipMemcpyHostToDevice);

	MulMatriz<<<dimGrid, dimBlock>>>(d_m1, d_m2, d_mr, fil1, col1, fil2, col2); //Ejecución del kernel
	hipMemcpy(h_mr, d_mr, sizer, hipMemcpyDeviceToHost); 

	EscribirMatriz(fil1, col1, h_m1);
	EscribirMatriz(fil2, col2, h_mr);
	EscribirMatriz(fil1, col2, h_mr);

	hipFree(d_m1);
	hipFree(d_m2);
	hipFree(d_mr);
	free(h_m1);
	free(h_m2);
	free(h_mr);

  	t_fin = clock(); //Fin reloj ------------------------
  	secs = (double)(t_fin - t_ini) / CLOCKS_PER_SEC;
  	printf("Tiempo de ejecucion: %.16g milisegundos\n", secs * 1000.0);
  	
  	return 0;
}
