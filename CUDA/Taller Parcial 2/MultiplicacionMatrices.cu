#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

__global__ void MulMatriz(float *m1, float *m2, float *mr, int fil1, int col1,int fil2, int col2)
{

	int i = blockIdx.y*blockDim.y + threadIdx.y;
	int j = blockIdx.x*blockDim.x + threadIdx.x;
	int valor = 0;

	for (int k=0; k<(blockDim.y+col1-1)/blockDim.y; k++) {
		for (int n=0; n<blockDim.y; ++n)
	    	if ((k*blockDim.y+n<col1 && i<fil1) && (k*blockDim.y+n<fil2 && j<col2))
	      		valor += m1[i*col1+k*blockDim.y+n]*m2[(k*blockDim.y+n)*col2+j];
	}

	if (i<fil1 && j<col2)
		mr[i*col2+j] = valor;

}


__host__
void LeerMatriz(float* m1, float* m2, FILE* file, int fil1, int fil2, int col1, int col2){
	
	for(int i=0; i<fil1*col1; i++){
		fscanf(file, "%f", &m1[i]);
    }

	for(int i=0; i<fil2*col2; i++){
		fscanf(file, "%f", &m2[i]);
    }

	fclose(file);
}


int main()
{
	//Inicia reloj ------------------------
	clock_t t_ini, t_fin;
  	double secs;
  	t_ini = clock();
  	//-------------------------------------

	int fil1, col1, fil2, col2;
	float *h_m1, *h_m2, *h_mr;
	float *d_m1, *d_m2, *d_mr;

	FILE *archivo;
	archivo = fopen("input.txt", "r");
    fscanf(archivo, "%d %d", &fil1, &col1);
    fscanf(archivo, "%d %d", &fil2, &col2);

	int size1 = fil1*col1*sizeof(float); //tamaño en bits de cada matriz
	int size2 = fil2*col2*sizeof(float);
	int sizer = fil1*col2*sizeof(float);

	h_m1 = (float*)malloc(size1);
	h_m2 = (float*)malloc(size2);
	h_mr = (float*)malloc(sizer);
	hipMalloc(&d_m1, size1);
	hipMalloc(&d_m2, size2);
	hipMalloc(&d_mr, sizer);

	int blockSize = 32;
	dim3 dimBlock(blockSize, blockSize, 1);
	dim3 dimGrid(ceil(col1/float(blockSize)), ceil(col1/float(blockSize)), 1);

	LeerMatriz(h_m1, h_m2, archivo, fil1, fil2, col1, col2);

	//Imprimir resultados------------------
	printf("matriz 1: ----------------------\n"); 
	for(int i=0; i<fil1; i++){
		for(int j=0; j<col1; j++){
			printf("%f ", h_m1[i*col1+j]);
		}
		printf("\n"); 
	}

	printf("matriz 2: ----------------------\n"); 
	for(int i=0; i<fil2; i++){
		for(int j=0; j<col2; j++){
			printf("%f ", h_m2[i*col2+j]);
		}
		printf("\n"); 
	}	

	printf("\nmatriz resultado: ----------------------\n"); 

	hipMemcpy(d_m1, h_m1, size1, hipMemcpyHostToDevice);
	hipMemcpy(d_m2, h_m2, size2, hipMemcpyHostToDevice);

	MulMatriz<<<dimGrid, dimBlock>>>(d_m1, d_m2, d_mr, fil1, col1, fil2, col2); //Ejecución del kernel
	hipMemcpy(h_mr, d_mr, sizer, hipMemcpyDeviceToHost); //Copia de datos al host
	
	//Imprimir resultados------------------
	for(int i=0; i<fil1; i++){
		for(int j=0; j<col2; j++){
			printf("%f ", h_mr[i*col2+j]);
		}
		printf("\n"); 
	}	
	//-------------------------------------

	hipFree(d_m1);
	hipFree(d_m2);
	hipFree(d_mr);
	free(h_m1);
	free(h_m2);
	free(h_mr);

	//Fin reloj ------------------------
  	t_fin = clock();
  	secs = (double)(t_fin - t_ini) / CLOCKS_PER_SEC;
  	printf("Tiempo de ejecucion: %.16g milisegundos\n", secs * 1000.0);
  	
  	return 0;
}
